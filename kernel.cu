#include "hip/hip_runtime.h"
﻿#ifndef CUDACC
#define CUDACC
#endif
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <conio.h>



double det(double* arr, int N) //Перемножаем элементы на главной диагонали , получаем определитель
{
    double d = 1.0;
    for (int i = 0; i < N; i++)
        d *= arr[i * N + i];
    return d;
}

__global__ void test(double* arr, int N)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x) / N;
    int j;
    double kof;
    for (j = 0; j < N; j++)
    {
        if (i >= j && i < N - 1)
        {
            kof = arr[(i + 1) * N + j] / arr[j * N + j];
            int g = (blockIdx.x * blockDim.x + threadIdx.x) % N;
            if (g < N)
            {
                arr[(i + 1) * N + g] -= kof * arr[j * N + g];
            }
        }
    }

}

__host__ int main()
{
    int N;
    printf("Enrer size of matrix N = ");
    scanf_s("%i", &N);
    int SizeMatr = N * N;//Размер матрицы
    int SizeInByte = SizeMatr * sizeof(double);//Память, необходимая для массива на GPU 
    double* pMatr = new double[SizeMatr];//Выделяем память под массив

    //Заполняем матрицу случайными числами и выводим 
    srand(time(NULL));
    for (int i = 0; i < SizeMatr; i++)
    {
        pMatr[i] = 1 + rand() % 9;
    }

    printf("\n");
    //for (int i = 0; i < SizeMatr; i++)
    //{
    //    printf("%0.2f ", pMatr[i]);
    //    if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    //}
    //printf("\n");


    double* pMatr_GPU;

    float start2 = clock();

    hipMalloc((void**)&pMatr_GPU, SizeInByte);//Выделяем память под массив на GPU
    hipMemcpy(pMatr_GPU, pMatr, SizeInByte, hipMemcpyHostToDevice);//Копируем значения матрицы на GPU 

    int gridsize = ((N * N) / 1024) + 1;
    int blocksize = 1024;

    //Инициализируем переменные для замера времени работы
    float recording;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    test << <  gridsize, blocksize >> > (pMatr_GPU, N); // вызов функции для изменения матрицы 

    float end = clock();
    //Получаем время работы
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&recording, start, stop);

    hipDeviceSynchronize();//Синхронизируем потоки

    hipMemcpy(pMatr, pMatr_GPU, SizeInByte, hipMemcpyDeviceToHost);//Копируем новую матрицу с GPU обратно на CPU
    printf("\n");
    //for (int i = 0; i < SizeMatr; i++)  //выводим измененную матрицу
    //{
    //    printf("%0.2f ", pMatr[i]);
    //    if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    //}
    //printf("\n");

    printf("\ndet A = %.2f \n", det(pMatr, N));//Выводим определитель
    if (recording > 0) printf("Time of execution =  %.2f\n", recording);
    else printf("Time working =  %.2f\n", end - start2);

    hipFree(pMatr_GPU);//Освобождаем память

    return 0;
}
